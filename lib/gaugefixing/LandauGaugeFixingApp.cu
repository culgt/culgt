#include "hip/hip_runtime.h"
/**
 *
 *  Created on: Apr 15, 2014
 *      Author: vogt
 */

#include "lattice/parameterization_types/ParameterizationMediatorSU2_Vector4_Real8.h"
#include "lattice/parameterization_types/ParameterizationMediatorSU3_Vector4_Real18.h"
#include "lattice/parameterization_types/ParameterizationMediatorSU3_Vector2_Real18.h"
#include "lattice/parameterization_types/ParameterizationMediatorSU3_Real12_Real18.h"
#include "application/GaugeConfigurationIteratingApplication.h"
#include "cuLGT1legacy/SiteIndex.hxx"
#include "lattice/configuration_patterns/GPUPatternTimesliceParityPriority.h"
#include "lattice/LinkFile.h"
#include "lattice/LatticeDimension.h"
#include "lattice/LinkFileVogt.h"
#include "lattice/LinkFileHirep.h"
#include "lattice/LinkFileHeaderOnly.h"
#include "observables/PlaquetteAverage.h"
#include "lattice/GlobalLink.h"
#include "gaugefixing/LandauGaugeFixing.h"
#include "util/rng/PhiloxWrapper.h"
#include "version.h"

#if __cplusplus >= 201103L
// this fixes a strange error in boost/lexical_cast.hpp where it needs std::pow( double, int )
#include <cmath>
double std::pow( double d, int i )
{
	return std::pow( d, (double)i );
}
#endif

namespace culgt
{

#ifdef DOUBLEPRECISION
typedef double REAL;
#else
typedef float REAL;
#endif


#if CULGT_SUN == 2

typedef SU2Vector4<REAL> PARAMTYPE;
typedef LocalLink<SU2Vector4<REAL> > LOCALLINK;

#else

#ifdef DOUBLEPRECISION
typedef SU3Vector2<REAL> PARAMTYPE;
#else
typedef SU3Vector4<REAL> PARAMTYPE;
#endif
typedef LocalLink<SUNRealFull<3,REAL> > LOCALLINK;

#endif


#ifdef CULGT_USE_TIMESLICE_PATTERN
typedef SiteIndex<4,TIMESLICE_SPLIT> SITE;
typedef GPUPatternTimesliceParityPriority<SITE,PARAMTYPE> PATTERNTYPE;
#else
typedef SiteIndex<4,FULL_SPLIT> SITE;
typedef GPUPatternParityPriority<SITE,PARAMTYPE> PATTERNTYPE;
#endif

typedef GlobalLink<PATTERNTYPE,true> GLOBALLINK;
typedef PhiloxWrapper<REAL> RNG;

#ifdef CULGT_FILETYPE_VOGT
typedef LinkFileVogt<PATTERNTYPE,REAL> FILETYPE;
#elif CULGT_FILETYPE_HIREP
typedef LinkFileHirep<PATTERNTYPE,REAL> FILETYPE;
#else
typedef LinkFileHeaderOnly<PATTERNTYPE,REAL> FILETYPE;
#endif

#ifdef CULGT_FILETYPE_VOGT_OUT
typedef LinkFileVogt<PATTERNTYPE,REAL> FILETYPEOUT;
#elif CULGT_FILETYPE_HIREP_OUT
typedef LinkFileHirep<PATTERNTYPE,REAL> FILETYPEOUT;
#elif CULGT_FILETYPE_HEADERONLY_OUT
typedef LinkFileHeaderOnly<PATTERNTYPE,REAL> FILETYPEOUT;
#else
typedef FILETYPE FILETYPEOUT;
#endif

/*
 *
 */
class LandauGaugeFixingApp: public GaugeConfigurationIteratingApplication<PATTERNTYPE,FILETYPE,FILETYPEOUT>
{
public:
	LandauGaugeFixingApp( const LatticeDimension<PATTERNTYPE::SITETYPE::Ndim> dim, FileIterator fileiterator, ProgramOptions* programOptions ) : GaugeConfigurationIteratingApplication<PATTERNTYPE,FILETYPE,FILETYPEOUT>(  dim, fileiterator, programOptions ), plaquette( configuration.getDevicePointer(), dimension )
	{
		programOptions->addOption( settings.getGaugeOptions() );

		boost::program_options::options_description gaugeOptions("Gaugefixing options");
		gaugeOptions.add_options()
				("sethot", boost::program_options::value<bool>(&sethot)->default_value(false), "start from a random gauge field")
				("fappendix", boost::program_options::value<string>(&fileAppendix)->default_value("gaugefixed_"), "file appendix (append after basename when writing)");

		programOptions->addOption( gaugeOptions );
		landau = new LandauGaugefixing<PATTERNTYPE,LOCALLINK>( configuration.getDevicePointer(), dimension, programOptions->getSeed() );
	}
private:
	GaugeSettings settings;

	void setup()
	{
		landau->orstepsAutoTune<RNG>(1.5, 50);
		landau->microcanonicalAutoTune<RNG>( 50 );
		landau->sastepsAutoTune<RNG>(1., 20);
	}

	void teardown()
	{
	}

	void iterate()
	{
		if( sethot )
		{
			std::cout << "Using a hot (random) lattice" << std::endl;
			configuration.setHotOnDevice<RNG>( programOptions->getSeed(), RNG::getNextCounter());
			CUDA_LAST_ERROR( "setHotOnDevice ");
			fix();
		}
		else
		{
			if( loadToDevice() )
			{
				fix();
				saveFromDevice( fileAppendix );
			}
		}
	};

	void fix()
	{
		std::cout << "Plaquette before: " << std::setprecision(12) << plaquette.getPlaquette() << std::endl;
		landau->fix( settings );
		std::cout << "Plaquette after: " << std::setprecision(12) << plaquette.getPlaquette() << std::endl;
	}

	PlaquetteAverage<PATTERNTYPE,LOCALLINK> plaquette;
	LandauGaugefixing<PATTERNTYPE,LOCALLINK>* landau;
	string fileAppendix;
	bool sethot;
};

} /* namespace culgt */


using namespace culgt;

#if BOOST_VERSION < 105300
int main( int argc, char* argv[] )
#else
int main( const int argc, const char* argv[] )
#endif
{
	std::cout << "cuLGT Version " << CULGT_VERSION << std::endl;
	LandauGaugeFixingApp::main<LandauGaugeFixingApp>( argc, argv );
}
