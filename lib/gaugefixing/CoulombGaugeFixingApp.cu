#include "hip/hip_runtime.h"
/**
 *
 *  Created on: Apr 15, 2014
 *      Author: vogt
 */

#include "lattice/parameterization_types/ParameterizationMediatorSU2_Vector4_Real8.h"
#include "lattice/parameterization_types/ParameterizationMediatorSU3_Vector4_Real18.h"
#include "lattice/parameterization_types/ParameterizationMediatorSU3_Vector2_Real18.h"
#include "application/GaugeConfigurationIteratingApplication.h"
#include "lattice/site_indexing/SiteIndex.h"
#include "lattice/configuration_patterns/GPUPatternTimesliceParityPriority.h"
#include "lattice/LinkFile.h"
#include "lattice/LatticeDimension.h"
#include "lattice/LinkFileVogt.h"
#include "lattice/LinkFileHirep.h"
#include "lattice/LinkFileHeaderOnly.h"
#include "lattice/filetypes/LinkFileILDG.h"
#include "observables/PlaquetteAverage.h"
#include "lattice/GlobalLink.h"
#include "gaugefixing/CoulombGaugeFixing.h"
#include "util/rng/PhiloxWrapper.h"
#include "gaugefixing/RandomGaugeTrafo.h"
#include "gaugefixing/GaugeSettings.h"
//#include "lattice/parameterization_types/SUNRealFull.h"
#include "version.h"

#if __cplusplus >= 201103L
// this fixes a strange error in boost/lexical_cast.hpp where it needs std::pow( double, int )
#include <cmath>
double std::pow( double d, int i )
{
	return std::pow( d, (double)i );
}
#endif

namespace culgt
{
#ifdef DOUBLEPRECISION
typedef double REAL;
#else
typedef float REAL;
#endif


#if CULGT_SUN == 2

typedef SU2Vector4<REAL> PARAMTYPE;
typedef LocalLink<SU2Vector4<REAL> > LOCALLINK;

#else

#ifdef DOUBLEPRECISION
typedef SU3Vector2<REAL> PARAMTYPE;
#else
typedef SU3Vector4<REAL> PARAMTYPE;
#endif
typedef LocalLink<SUNRealFull<3,REAL> > LOCALLINK;

#endif


typedef SiteIndex<4,TIMESLICE_SPLIT> SITE;
typedef GPUPatternTimesliceParityPriority<SITE,PARAMTYPE> PATTERNTYPE;
typedef GlobalLink<PATTERNTYPE,true> GLOBALLINK;
//typedef GlobalLink<PATTERNTYPE::TIMESLICE_PATTERNTYPE,true> GLOBALLINKTIMESLICE;
typedef PhiloxWrapper<REAL> RNG;

#ifdef CULGT_FILETYPE_VOGT
typedef LinkFileVogt<PATTERNTYPE,REAL> FILETYPE;
#elif CULGT_FILETYPE_HIREP
typedef LinkFileHirep<PATTERNTYPE,REAL> FILETYPE;
#elif CULGT_FILETYPE_ILDG
typedef LinkFileILDG<PATTERNTYPE,REAL> FILETYPE;
#else
typedef LinkFileHeaderOnly<PATTERNTYPE,REAL> FILETYPE;
#endif

#ifdef CULGT_FILETYPE_VOGT_OUT
typedef LinkFileVogt<PATTERNTYPE,REAL> FILETYPEOUT;
#elif CULGT_FILETYPE_HIREP_OUT
typedef LinkFileHirep<PATTERNTYPE,REAL> FILETYPEOUT;
#elif CULGT_FILETYPE_HEADERONLY_OUT
typedef LinkFileHeaderOnly<PATTERNTYPE,REAL> FILETYPEOUT;
#else
typedef FILETYPE FILETYPEOUT;
#endif

/*
 *
 */
class CoulombGaugeFixingApp: public GaugeConfigurationIteratingApplication<PATTERNTYPE,FILETYPE,FILETYPEOUT>
{
public:
	CoulombGaugeFixingApp( const LatticeDimension<PATTERNTYPE::SITETYPE::NDIM> dim, FileIterator fileiterator, ProgramOptions* programOptions ) : GaugeConfigurationIteratingApplication<PATTERNTYPE,FILETYPE,FILETYPEOUT>(  dim, fileiterator, programOptions ), plaquette( configuration.getDevicePointer(), dimension )
	{
		programOptions->addOption( settings.getGaugeOptions() );

		boost::program_options::options_description gaugeOptions;
		gaugeOptions.add_options()
				("sethot", boost::program_options::value<bool>(&sethot)->default_value(false), "start from a random gauge field")
				("fappendix", boost::program_options::value<string>(&fileAppendix)->default_value("gaugefixed_"), "file appendix (append after basename when writing)")
				("timeslice", boost::program_options::value<int>(&fixSlice)->default_value(-1), "fix only specific timeslice (-1 = fix all)");

		programOptions->addOption( gaugeOptions );


		coulomb = new CoulombGaugefixing<PATTERNTYPE::TIMESLICE_PATTERNTYPE,LOCALLINK>( configuration.getDevicePointer( 0 ), configuration.getDevicePointer( dim.getDimension(0)-1 ), dim.getDimensionTimeslice(), programOptions->getSeed() );
	}
private:
	GaugeSettings settings;

	void setup()
	{
		coulomb->orstepsAutoTune<RNG>(1.5, 200);
//		coulomb->cornellAutoTune<RNG>(.5, 200);
		coulomb->sastepsAutoTune<RNG>(.5, 200);
		coulomb->microcanonicalAutoTune<RNG>( 200 );
	}

	void teardown()
	{
	}

	void iterate()
	{
		if( sethot )
		{
			std::cout << "Using a hot (random) lattice" << std::endl;
			configuration.setHotOnDevice<RNG>( programOptions->getSeed(), RNG::getNextCounter());
			CUDA_LAST_ERROR( "setHotOnDevice ");
			if( fixSlice == -1)
				fix();
			else
				fix( fixSlice );
		}
		else
		{
			if( loadToDevice() )
			{
				if( fixSlice == -1)
					fix();
				else
					fix( fixSlice );
				saveFromDevice( fileAppendix );
			}
		}
	};

	void fix( int t )
	{
		int tDown = (t == 0)?(dimension.getDimension(0)-1):t-1;
		std::cout << "Timeslice t = " << t << " (" << tDown << ")"<< std::endl;
		coulomb->setTimeslice( configuration.getDevicePointer( t ), configuration.getDevicePointer(tDown) );
		coulomb->fix( settings );
	}

	void fix()
	{
		std::cout << "Plaquette before: " << std::setprecision(12) << plaquette.getPlaquette() << std::endl;
//
//		int t = 0;
//		int tDown = (t == 0)?(dimension.getDimension(0)-1):t-1;
//		std::cout << "Timeslice t = " << t << " (" << tDown << ")"<< std::endl;
//		coulomb->setTimeslice( configuration.getDevicePointer( t ), configuration.getDevicePointer(tDown) );
//
//		GaugeStats stats = coulomb->getGaugeStats();
//		std::cout << 0 << " \t" << stats.getGff() << " \t" << stats.getPrecision() << std::endl;
////		stats = coulomb->getGaugeStats( GAUGEFIELD_LOGARITHMIC );
////		std::cout << 0 << " \t" << stats.getGff() << " \t" << stats.getPrecision() << std::endl;
//
//		coulomb->randomTrafo();
//
//		coulomb->fix( settings );
////		for( int j = 0; j < 10000; j++ )
////		{
////			int iter = 100;
////			for( int i = 0; i < iter; i++ )
////			{
////				coulomb->runCornell( .1, 5 );
//////				coulomb->runOverrelaxation( 1.5 );
////				CUDA_LAST_ERROR( "Cornell ");
////			}
////				coulomb->reproject();
////
////
////			stats = coulomb->getGaugeStats();
////			std::cout << 0 << " \t" << stats.getGff() << " \t" << stats.getPrecision() << std::endl;
////			if( stats.getPrecision() < settings.getPrecision() ) break;
////			stats = coulomb->getGaugeStats( GAUGEFIELD_LOGARITHMIC );
////			std::cout << 0 << " \t" << stats.getGff() << " \t" << stats.getPrecision() << std::endl;
////		}
//		std::cout << "Plaquette: " << std::setprecision(12) << plaquette.getPlaquette() << std::endl;
//		exit( 1 );

		RunInfo info;

		for( int t = 0; t < dimension.getDimension(0); t++ )
		{
			fix(t);
		}

		std::cout << "Plaquette after: " << std::setprecision(12) << plaquette.getPlaquette() << std::endl;
	}

	CoulombGaugefixing<PATTERNTYPE::TIMESLICE_PATTERNTYPE,LOCALLINK>* coulomb;
	PlaquetteAverage<PATTERNTYPE,LOCALLINK> plaquette;
	string fileAppendix;
	int fixSlice;
	bool sethot;
};

} /* namespace culgt */


using namespace culgt;

#if BOOST_VERSION < 105300
int main( int argc, char* argv[] )
#else
int main( const int argc, const char* argv[] )
#endif
{
	std::cout << "cuLGT Version " << CULGT_VERSION << std::endl;
	CoulombGaugeFixingApp::main<CoulombGaugeFixingApp>( argc, argv );
}
