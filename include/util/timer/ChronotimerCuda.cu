/**
 *
 * This is more ore less a hack to allow easy switching between both implementations...
 *
 */



#include "Chronotimer.h"

Chronotimer::Chronotimer()
{
	hipEventCreate(&cudastart);
	hipEventCreate(&cudastop);
	resetted = true;
	running = false;
}

void Chronotimer::start()
{
	if( !running )
	{
		hipEventRecord(cudastart, 0);
		running = true;
	}
}

void Chronotimer::stop()
{
	hipEventRecord(cudastop, 0);
	hipEventSynchronize(cudastop);
	running = false;
}

void Chronotimer::reset()
{
	resetted = true;
	running = false;
}

double Chronotimer::getTime()
{
	hipEventElapsedTime(&cudatime, cudastart, cudastop);
	return (double)cudatime/1000.;
}

double Chronotimer::getElapsedSeconds()
{
	return 0;
}
