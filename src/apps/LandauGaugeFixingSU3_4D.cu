#include "hip/hip_runtime.h"
/*
 * test_gaugefixing.cpp
 *
 *  Created on: Apr 18, 2012
 *      Author: vogt
 */

#include <iostream>
#include <math.h>
#include <sstream>
#include <malloc.h>
#include "../lattice/gaugefixing/GaugeFixingSubgroupStep.hxx"
#include "../lattice/gaugefixing/GaugeFixingStats.hxx"
#include "../lattice/gaugefixing/overrelaxation/OrUpdate.hxx"
#include "../lattice/access_pattern/StandardPattern.hxx"
#include "../lattice/access_pattern/GpuCoulombPattern.hxx"
#include "../lattice/access_pattern/GpuLandauPattern.hxx"
#include "../lattice/SiteCoord.hxx"
#include "../lattice/SiteIndex.hxx"
#include "../lattice/Link.hxx"
#include "../lattice/SU3.hxx"
#include "../lattice/Matrix.hxx"
#include "../lattice/LinkFile.hxx"
#include "../lattice/gaugefixing/overrelaxation/OrSubgroupStep.hxx"
#include "../util/timer/Chronotimer.h"
#include "../lattice/filetypes/FileHeaderOnly.hxx"
#include "../lattice/filetypes/FilePlain.hxx"
#include "../lattice/filetypes/FileVogt.hxx"
#include "../lattice/filetypes/filetype_typedefs.h"
#include <boost/program_options/parsers.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/program_options/options_description.hpp>


using namespace std;

const lat_dim_t Ndim = 4;
const short Nc = 3;

#ifdef _X_
const lat_coord_t Nx = _X_;
#else
#error "Define X (the lattice size in x-direction)"
#endif
#ifdef _Y_
const lat_coord_t Ny = _Y_;
#else
const lat_coord_t Ny = _X_;
bool warnY = true; // TODO print the warning
#endif
#ifdef _Z_
const lat_coord_t Nz = _Z_;
#else
const lat_coord_t Nz = _X_;
bool warnZ = true;
#endif
#ifdef _T_
const lat_coord_t Nt = _T_;
#else
#error "Define T (the lattice size in t-direction)"
#endif


// boost program options setup
boost::program_options::variables_map options_vm;
boost::program_options::options_description options_desc("Allowed options");

// parameters from command line or config file
int nconf;
long seed; // TODO check datatype
int orMaxIter;
int orCheckPrec;
float orParameter;
float orPrecision;
int saSteps;
float saMin;
float saMax;
int gaugeCopies;
string fileEnding;
string postFixLabel;
string fileBasename;
int fileStartnumber;
int fileNumberformat;
string configFile;
bool noRandomTrafo;
FileType fileType;


// lattice setup
const lat_coord_t size[Ndim] = {Nt,Nx,Ny,Nz};
__constant__ lat_coord_t dSize[Ndim] = {Nt,Nx,Ny,Nz};
const int arraySize = Nt*Nx*Ny*Nz*Ndim*Nc*Nc*2;
const int timesliceArraySize = Nx*Ny*Nz*Ndim*Nc*Nc*2;

typedef StandardPattern<SiteCoord<Ndim,false>,Ndim,Nc> Standard;
typedef GpuLandauPattern< SiteCoord<Ndim,true>,Ndim,Nc> Gpu;


typedef Link<Gpu,SiteCoord<Ndim,true>,Ndim,Nc> TLink;


__device__ inline Real cuFabs( Real a )
{
	return (a>0)?(a):(-a);
}

void initNeighbourTable( lat_index_t* nnt )
{
	const lat_coord_t size[Ndim] = {Nt,Nx,Ny,Nz};
	SiteIndex<4,true> s(size);
	s.calculateNeighbourTable( nnt );
}


__global__ void projectSU3( Real* U )
{
	const lat_coord_t size[Ndim] = {Nt,Nx,Ny,Nz};
	SiteCoord<4,true> s(size);
	int site = blockIdx.x * blockDim.x + threadIdx.x;

	s.setLatticeIndex( site );

	for( int mu = 0; mu < 4; mu++ )
	{
		TLink linkUp( U, s, mu );
		SU3<TLink> globUp( linkUp );

		globUp.projectSU3();
	}
}


__global__ void __launch_bounds__(256,4) orStep( Real* U, lat_index_t* nn, bool parity, float orParameter )
{
	typedef GpuLandauPattern< SiteIndex<Ndim,true>,Ndim,Nc> GpuIndex;
	typedef Link<GpuIndex,SiteIndex<Ndim,true>,Ndim,Nc> TLinkIndex;

	const lat_coord_t size[Ndim] = {Nt,Nx,Ny,Nz};
	SiteIndex<4,true> s(size);
	s.nn = nn;

	const bool updown = threadIdx.x / 128;
	const short mu = (threadIdx.x % 128) / 32;
	const short id = (threadIdx.x % 128) % 32;

	int site = blockIdx.x * blockDim.x/8 + id;
	if( parity == 1 ) site += s.getLatticeSize()/2;

	s.setLatticeIndex( site );
	if( updown==1 )
	{
		s.setNeighbour(mu,false);
	}

//	if(id == 0) printf("bin in or\n");

	Matrix<complex,Nc> locMat;
	SU3<Matrix<complex,Nc> > locU(locMat);

	TLinkIndex link( U, s, mu );

	SU3<TLinkIndex> globU( link );

	// make link local
	locU.assignWithoutThirdLine(globU);
	locU.reconstructThirdLine();

	// define the update algorithm
	OrUpdate overrelax( orParameter );
	GaugeFixingSubgroupStep<SU3<Matrix<complex,Nc> >, OrUpdate, LANDAU> subgroupStep( &locU, overrelax, id, mu, updown );

	// do the subgroup iteration
	SU3<Matrix<complex,Nc> >::perSubgroup( subgroupStep );

	// copy link back
	globU.assignWithoutThirdLine(locU);
	
	// project back
	//globU.projectSU3withoutThirdRow();
}




Real calculatePolyakovLoopAverage( Real *U )
{
	Matrix<complex,3> tempMat;
	SU3<Matrix<complex,3> > temp( tempMat );
	Matrix<complex,3> temp2Mat;
	SU3<Matrix<complex,3> > temp2( temp2Mat );

	SiteCoord<Ndim,true> s( size );

	complex result(0,0);

	for( s[1] = 0; s[1] < s.size[1]; s[1]++ )
	{
		for( s[2] = 0; s[2] < s.size[2]; s[2]++ )
		{
			for( s[3] = 0; s[3] < s.size[3]; s[3]++ )
			{
				temp.identity();
				temp2.zero();

				for( s[0] = 0; s[0] < s.size[0]; s[0]++ )
				{

					TLink link( U, s, 0 );
					SU3<TLink> globU( link );

					temp2 = temp2 + temp*globU;

					temp = temp2;
					temp2.zero();
				}
				result += temp.trace();
			}
		}
	}

	return sqrt(result.x*result.x+result.y*result.y) / (Real)(s.getLatticeSizeTimeslice()*Nc);
}






int main(int argc, char* argv[])
{
	// read parameters (command line or given config file)
	options_desc.add_options()
		("help", "produce help message")
		("nconf,m", boost::program_options::value<int>(&nconf)->default_value(1), "how many files to gaugefix")
		("ormaxiter", boost::program_options::value<int>(&orMaxIter)->default_value(1000), "Max. number of OR iterations")
		("seed", boost::program_options::value<long>(&seed)->default_value(1), "RNG seed")
		("sasteps", boost::program_options::value<int>(&saSteps)->default_value(1000), "number of SA steps")
		("samin", boost::program_options::value<float>(&saMin)->default_value(.01), "min. SA temperature")
		("samax", boost::program_options::value<float>(&saMax)->default_value(.4), "max. SA temperature")
		("orparameter", boost::program_options::value<float>(&orParameter)->default_value(1.7), "OR parameter")
		("orprecision", boost::program_options::value<float>(&orPrecision)->default_value(1E-7), "OR precision (dmuAmu)")
		("orcheckprecision", boost::program_options::value<int>(&orCheckPrec)->default_value(100), "how often to check the gauge precision")
		("gaugecopies", boost::program_options::value<int>(&gaugeCopies)->default_value(1), "Number of gauge copies")
		("ending", boost::program_options::value<string>(&fileEnding)->default_value(".vogt"), "file ending to append to basename (default: .vogt)")
		("postfixlabel", boost::program_options::value<string>(&postFixLabel)->default_value("_Landau"), "label to append to basename after fixing the gauge and before storing it (default _Landau)")
		("basename", boost::program_options::value<string>(&fileBasename), "file basename (part before numbering starts)")
		("startnumber", boost::program_options::value<int>(&fileStartnumber)->default_value(0), "file index number to start from (startnumber, ..., startnumber+nconf-1")
		("numberformat", boost::program_options::value<int>(&fileNumberformat)->default_value(1), "number format for file index: 1 = (0,1,2,...,10,11), 2 = (00,01,...), 3 = (000,001,...),...")
		("filetype", boost::program_options::value<FileType>(&fileType), "type of configuration (PLAIN, HEADERONLY, VOGT)")
		("config-file", boost::program_options::value<string>(&configFile), "config file (command line arguments overwrite config file settings)")

		("norandomtrafo", boost::program_options::value<bool>(&noRandomTrafo)->default_value(false), "no random gauge trafo" )
		;

	boost::program_options::positional_options_description options_p;
	options_p.add("config-file", -1);

	boost::program_options::store(boost::program_options::command_line_parser(argc, argv).
			options(options_desc).positional(options_p).run(), options_vm);
	boost::program_options::notify(options_vm);

	ifstream cfg( configFile.c_str() );
	boost::program_options::store(boost::program_options::parse_config_file( cfg, options_desc), options_vm);
	boost::program_options::notify(options_vm);

	if (options_vm.count("help")) {
		cout << "Usage: " << argv[0] << " [options] [config-file]" << endl;
		cout << options_desc << "\n";
		return 1;
	}












	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	printf("\nDevice %d: \"%s\"\n", 0, deviceProp.name);
	printf("CUDA Capability Major/Minor version number:    %d.%d\n\n", deviceProp.major, deviceProp.minor);

	Chronotimer allTimer;
	allTimer.reset();

	SiteCoord<4,true> s(size);


	// TODO maybe we should choose the filetype on compile time
	LinkFile<FileHeaderOnly, Standard, Gpu, SiteCoord<4,true> > lfHeaderOnly;
	LinkFile<FileVogt, Standard, Gpu, SiteCoord<4,true> > lfVogt;
	LinkFile<FilePlain, Standard, Gpu, SiteCoord<4,true> > lfPlain;


	// allocate Memory
	// host memory for configuration
	Real* U = (Real*)malloc( arraySize*sizeof(Real) );

	// device memory for configuration
	Real* dU;
	hipMalloc( &dU, arraySize*sizeof(Real) );

	// host memory for the neighbour table
	lat_index_t* nn = (lat_index_t*)malloc( s.getLatticeSize()*(2*(Ndim))*sizeof(lat_index_t) );

	// device memory for the timeslice neighbour table
	lat_index_t *dNn;
	hipMalloc( &dNn, s.getLatticeSize()*(2*(Ndim))*sizeof( lat_index_t ) );



	// initialise the timeslice neighbour table
	initNeighbourTable( nn );
	// copy neighbour table to device
	hipMemcpy( dNn, nn, s.getLatticeSize()*(2*(Ndim))*sizeof( lat_index_t ), hipMemcpyHostToDevice );

	int threadsPerBlock = 32*8; // 32 sites are updated within a block (8 threads are needed per site)
	int numBlocks = s.getLatticeSize()/2/32; // // half of the lattice sites (a parity) are updated in a kernel call

	allTimer.start();

	hipFuncSetCacheConfig(reinterpret_cast<const void*>( orStep), hipFuncCachePreferL1 );
	
	// instantiate GaugeFixingStats object
	lat_coord_t *devicePointerToSize;
	hipGetSymbolAddress( (void**)&devicePointerToSize, "dSize" );
	GaugeFixingStats<Ndim,Nc,LANDAU> gaugeStats( dU, &size[0], devicePointerToSize );


	double totalKernelTime = 0;

	long totalStepNumber = 0;

	for( int i = fileStartnumber; i < fileStartnumber+nconf; i++ )
	{

		stringstream filename(stringstream::out);
		filename << fileBasename << setw( fileNumberformat ) << setfill( '0' ) << i << fileEnding;
//		filename << "/home/vogt/configs/STUDIENARBEIT/N32/config_n32t32beta570_sp" << setw( 4 ) << setfill( '0' ) << i << ".vogt";
		cout << "loading " << filename.str() << " as " << fileType << endl;

		bool loadOk;

		switch( fileType )
		{
		case VOGT:
			loadOk = lfVogt.load( s, filename.str(), U );
			break;
		case PLAIN:
			loadOk = lfPlain.load( s, filename.str(), U );
			break;
		case HEADERONLY:
			loadOk = lfHeaderOnly.load( s, filename.str(), U );
			break;
		default:
			cout << "Filetype not set to a known value. Exiting";
			exit(1);
		}

		if( !loadOk )
		{
			cout << "Error while loading. Trying next file." << endl;
			break;
		}
		else
		{
			cout << "File loaded." << endl;
		}
//		Real polBefore = calculatePolyakovLoopAverage( U );

		// copying configuration ...
		hipMemcpy( dU, U, arraySize*sizeof(Real), hipMemcpyHostToDevice );

		// calculate and print the gauge quality
		printf( "i:\t\tgff:\t\tdA:\n");
		gaugeStats.generateGaugeQuality();

		Chronotimer kernelTimer;
		kernelTimer.reset();
		kernelTimer.start();
		for( int i = 0; i < orMaxIter; i++ )
		{
			orStep<<<numBlocks,threadsPerBlock>>>(dU, dNn, 0, orParameter );
			orStep<<<numBlocks,threadsPerBlock>>>(dU, dNn, 1, orParameter );

			if( i % orCheckPrec == 0 )
			{
//				projectSU3<<<numBlocks*2,32>>>( dU );
		// check the current gauge quality
			gaugeStats.generateGaugeQuality();
			printf( "%d\t\t%1.10f\t\t%e\n", i, gaugeStats.getCurrentGff(), gaugeStats.getCurrentA() );

			if( gaugeStats.getCurrentA() < orPrecision ) break;
			}

			totalStepNumber++;
		}
		hipDeviceSynchronize();
		kernelTimer.stop();
		cout << "kernel time for config: " << kernelTimer.getTime() << " s"<< endl;
		totalKernelTime += kernelTimer.getTime();
		hipMemcpy( U, dU, arraySize*sizeof(Real), hipMemcpyDeviceToHost );

//		cout << "Polyakov loop: " << polBefore << " - " << calculatePolyakovLoopAverage( U ) << endl;
	}

	allTimer.stop();
	cout << "total time: " << allTimer.getTime() << " s" << endl;
	cout << "total kernel time: " << totalKernelTime << " s" << endl;

	cout << (double)((long)2253*(long)s.getLatticeSize()*(long)totalStepNumber)/totalKernelTime/1.0e9 << " GFlops at "
				<< (double)((long)192*(long)s.getLatticeSize()*(long)(totalStepNumber)*(long)sizeof(Real))/totalKernelTime/1.0e9 << "GB/s memory throughput." << endl;

}
