#include "hip/hip_runtime.h"
/*
 * test_gaugefixing.cpp
 *
 *  Created on: Apr 18, 2012
 *      Author: vogt
 */

#include <iostream>
#include <math.h>
#include <sstream>
#include <malloc.h>
#include "../lattice/gaugefixing/GaugeFixingSubgroupStep.hxx"
#include "../lattice/gaugefixing/overrelaxation/OrUpdate.hxx"
#include "../lattice/access_pattern/StandardPattern.hxx"
#include "../lattice/access_pattern/GpuCoulombPattern.hxx"
#include "../lattice/access_pattern/GpuLandauPattern.hxx"
#include "../lattice/SiteCoord.hxx"
#include "../lattice/SiteIndex.hxx"
#include "../lattice/Link.hxx"
#include "../lattice/SU3.hxx"
#include "../lattice/Matrix.hxx"
#include "../lattice/LinkFile.hxx"
#include "../lattice/gaugefixing/overrelaxation/OrSubgroupStep.hxx"
#include "../util/timer/Chronotimer.h"
#include "../lattice/filetypes/FileVogt.hxx"
#include "../util/datatype/lattice_typedefs.h"

using namespace std;

const lat_dim_t Ndim = 4;
const short Nc = 3;

#ifdef _X_
const lat_coord_t Nx = _X_;
#else
#error "Define X (the lattice size in x-direction)"
#endif
#ifdef _Y_
const lat_coord_t Ny = _Y_;
#else
const lat_coord_t Ny = _X_;
bool warnY = true; // TODO print the warning
#endif
#ifdef _Z_
const lat_coord_t Nz = _Z_;
#else
const lat_coord_t Nz = _X_;
bool warnZ = true;
#endif
#ifdef _T_
const lat_coord_t Nt = _T_;
#else
#error "Define T (the lattice size in t-direction)"
#endif

const lat_coord_t size[Ndim] = {Nt,Nx,Ny,Nz};
const int arraySize = Nt*Nx*Ny*Nz*Ndim*Nc*Nc*2;
const int timesliceArraySize = Nx*Ny*Nz*Ndim*Nc*Nc*2;

typedef GpuCoulombPattern<SiteCoord<Ndim,true>,Ndim,Nc> Gpu;
typedef StandardPattern<SiteCoord<Ndim,false>,Ndim,Nc> Standard;
typedef GpuLandauPattern< SiteCoord<Ndim-1,true>,Ndim-1,Nc> GpuTimeslice;


typedef Link<Gpu,SiteCoord<Ndim,true>,Ndim,Nc> TLink;
typedef Link<GpuTimeslice,SiteCoord<Ndim-1,true>,Ndim-1,Nc> TLink3;

void initNeighbourTable( lat_index_t* nnt )
{
	const lat_coord_t size[Ndim-1] = {Nx,Ny,Nz};
	SiteIndex<3,true> s(size);
	s.calculateNeighbourTable( nnt );
}


__device__ inline Real cuFabs( Real a )
{
	return (a>0)?(a):(-a);
}

__global__ void printGaugeQuality( Real* dGff, Real* dA )
{
	const lat_coord_t size[Ndim-1] = {Nx,Ny,Nz};
	SiteCoord<3,true> s(size);

	Real gff = 0;
	Real temp = 0;
	for( int i = 0; i < s.getLatticeSize(); i++ )
	{
		gff+= dGff[i];
		if( cuFabs(dA[i]) > temp ) temp = cuFabs(dA[i]);
	}

	printf( "gff: %E\t\tdA: %E\n", gff/Real(s.getLatticeSize())/3./3., temp );

}

__global__ void projectSU3( Real* U )
{
	const lat_coord_t size[Ndim-1] = {Nx,Ny,Nz};
	SiteCoord<3,true> s(size);
	int site = blockIdx.x * blockDim.x + threadIdx.x;

	s.setLatticeIndex( site );

	for( int mu = 0; mu < 4; mu++ )
	{
		TLink3 linkUp( U, s, mu );
		SU3<TLink3> globUp( linkUp );

		globUp.projectSU3();
	}
}

__global__ void generateGaugeQuality( Real *U, Real *dGff, Real *dA )
{
	const lat_coord_t size[Ndim-1] = {Nx,Ny,Nz};
	SiteCoord<3,true> s(size);
	int site = blockIdx.x * blockDim.x + threadIdx.x;

	Matrix<complex,Nc> locMatSum;
	SU3<Matrix<complex,Nc> > Sum(locMatSum);

	Sum.zero();

	// TODO check if there is a faster way to compute DELTA
	for( int mu = 1; mu < 4; mu++ )
	{
		s.setLatticeIndex( site );

		Matrix<complex,Nc> locMat;
		SU3<Matrix<complex,Nc> > temp(locMat);

		TLink3 linkUp( U, s, mu );
		SU3<TLink3> globUp( linkUp );

		temp.assignWithoutThirdLine( globUp );
		temp.reconstructThirdLine();
		Sum += temp;

		s.setNeighbour(mu-1,-1);
		TLink3 linkDw( U, s, mu );
		SU3<TLink3> globDw( linkDw );
		temp.assignWithoutThirdLine( globDw );
		temp.reconstructThirdLine();
		Sum -= temp;
	}

	Sum -= Sum.trace()/Real(3.);

	Matrix<complex,Nc> locMatSumHerm;
	SU3<Matrix<complex,Nc> > SumHerm(locMatSumHerm);
	SumHerm = Sum;
	SumHerm.hermitian();

	Sum -= SumHerm;

	Real prec = 0;
	for( int i = 0; i < 3; i++ )
	{
		for( int j = 0; j < 3; j++ )
		{
			prec += Sum.get(i,j).abs_squared();
		}
	}

	dA[site] = prec;


	s.setLatticeIndex( site );
	Real result = 0;


	Matrix<complex,Nc> locTemp;
	SU3<Matrix<complex,Nc> > temp(locTemp);
	for( int mu = 1; mu < 4; mu++ )
	{
		TLink3 linkUp( U, s, mu );
		SU3<TLink3> globUp( linkUp );
		temp.assignWithoutThirdLine( globUp ); // TODO don't load twice
		temp.reconstructThirdLine();
		result += temp.trace().x;
	}

	dGff[site] = result;
}



__global__ void __launch_bounds__(256,4) orStep( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, float orParameter )
{
	typedef GpuLandauPattern< SiteIndex<Ndim-1,true>,Ndim-1,Nc> GpuTimeslice_2;
	typedef Link<GpuTimeslice_2,SiteIndex<Ndim-1,true>,Ndim-1,Nc> TLink3_2;

	const lat_coord_t size[Ndim-1] = {Nx,Ny,Nz};
	SiteIndex<3,true> s(size);
	s.nn = nnt;

	const bool updown = threadIdx.x / 128;
	const short mu = (threadIdx.x % 128) / 32;
	const short id = (threadIdx.x % 128) % 32;

	int site = blockIdx.x * blockDim.x/8 + id;
	if( parity == 1 ) site += s.getLatticeSize()/2;

	s.setLatticeIndex( site );
	if( (mu!=0)&&(updown==1) )
	{
		s.setNeighbour(mu-1,0);
	}

	Matrix<complex,Nc> locMat;
	SU3<Matrix<complex,Nc> > locU(locMat);

	TLink3_2 link( ((mu==0)&&(updown==1))?(UtDw):(UtUp), s, mu );

	SU3<TLink3_2> globU( link );

	// make link local
	locU.assignWithoutThirdLine(globU);
	locU.reconstructThirdLine();

	// define the update algorithm
	OrUpdate overrelax( orParameter );
	GaugeFixingSubgroupStep<SU3<Matrix<complex,Nc> >, OrUpdate, COULOMB> subgroupStep( &locU, overrelax, id, mu, updown );

	// do the subgroup iteration
	SU3<Matrix<complex,Nc> >::perSubgroup( subgroupStep );

	// copy link back
	globU.assignWithoutThirdLine(locU);
}




Real calculatePolyakovLoopAverage( Real *U )
{
	Matrix<complex,3> tempMat;
	SU3<Matrix<complex,3> > temp( tempMat );
	Matrix<complex,3> temp2Mat;
	SU3<Matrix<complex,3> > temp2( temp2Mat );

	SiteCoord<Ndim,true> s( size );

	complex result(0,0);

	for( s[1] = 0; s[1] < s.size[1]; s[1]++ )
	{
		for( s[2] = 0; s[2] < s.size[2]; s[2]++ )
		{
			for( s[3] = 0; s[3] < s.size[3]; s[3]++ )
			{
				temp.identity();
				temp2.zero();

				for( s[0] = 0; s[0] < s.size[0]; s[0]++ )
				{

					TLink link( U, s, 0 );
					SU3<TLink> globU( link );

					temp2 = temp2 + temp*globU;

					temp = temp2;
					temp2.zero();
				}
				result += temp.trace();
			}
		}
	}

	return sqrt(result.x*result.x+result.y*result.y) / (Real)(s.getLatticeSizeTimeslice()*Nc);
}






int main(int argc, char* argv[])
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	printf("\nDevice %d: \"%s\"\n", 0, deviceProp.name);
	printf("CUDA Capability Major/Minor version number:    %d.%d\n\n", deviceProp.major, deviceProp.minor);

	Chronotimer allTimer;
	allTimer.reset();

	SiteCoord<4,true> s(size);
	LinkFile<FileVogt, Standard, Gpu, SiteCoord<4,true> > lf;


	// allocate Memory
	// host memory for configuration
	Real* U = (Real*)malloc( arraySize*sizeof(Real) );

	// device memory for timeslice t
	Real* dUtUp;
	hipMalloc( &dUtUp, timesliceArraySize*sizeof(Real) );

	// device memory for timeslice t-1
	Real* dUtDw;
	hipMalloc( &dUtDw, timesliceArraySize*sizeof(Real) );

	// device memory for collecting the parts of the gauge fixing functional and divA
	Real *dGff;
	hipMalloc( &dGff, s.getLatticeSizeTimeslice()*sizeof(Real) );
	Real *dA;
	hipMalloc( &dA, s.getLatticeSizeTimeslice()*sizeof(Real) );

	// host memory for the timeslice neighbour table
	lat_index_t* nnt = (lat_index_t*)malloc( s.getLatticeSizeTimeslice()*(2*(Ndim-1))*sizeof(lat_index_t) );

	// device memory for the timeslice neighbour table
	lat_index_t *dNnt;
	hipMalloc( &dNnt, s.getLatticeSizeTimeslice()*(2*(Ndim-1))*sizeof( lat_index_t ) );



	// initialise the timeslice neighbour table
	initNeighbourTable( nnt );
	// copy neighbour table to device
	hipMemcpy( dNnt, nnt, s.getLatticeSizeTimeslice()*(2*(Ndim-1))*sizeof( lat_index_t ), hipMemcpyHostToDevice );

	int threadsPerBlock = 32*8; // 32 sites are updated within a block (8 threads are needed per site)
	int numBlocks = s.getLatticeSizeTimeslice()/2/32; // // half of the lattice sites (a parity) are updated in a kernel call

	allTimer.start();

	hipFuncSetCacheConfig(reinterpret_cast<const void*>( orStep), hipFuncCachePreferL1 );

//	uint kernelTimer;
//	cutCreateTimer( &kernelTimer );

	double totalKernelTime = 0;

	for( int i = 0; i < 1; i++ )
	{

		stringstream filename(stringstream::out);
//		filename << "/home/vogt/configs/STUDIENARBEIT/N32/config_n32t32beta570_" << setw( 4 ) << setfill( '0' ) << i << ".vogt";
		filename << "/home/vogt/configs/STUDIENARBEIT/N32/config_n32t32beta570_sp" << setw( 4 ) << setfill( '0' ) << i << ".vogt";
//		filename << "/home/vogt/configs/STUDIENARBEIT/N16/config_n16t16beta570_sp" << setw( 4 ) << setfill( '0' ) << i << ".vogt.gf";

		bool loadOk = lf.load( s, filename.str(), U );

		if( !loadOk )
		{
			cout << "Error while loading. Trying next file." << endl;
			break;
		}
		else
		{
			cout << "File loaded." << endl;
		}

		for( int t = 0; t < s.size[0]; t++ )
		{
			int tDw = (t > 0)?(t-1):(s.size[0]-1); // calculating t-1 (periodic boundaries)

			// copying timeslice t ...
			hipMemcpy( dUtUp, &U[t*timesliceArraySize], timesliceArraySize*sizeof(Real), hipMemcpyHostToDevice );
			// ... and t-1 to device
			hipMemcpy( dUtDw, &U[tDw*timesliceArraySize], timesliceArraySize*sizeof(Real), hipMemcpyHostToDevice );
			// TODO it is not necessary to copy the (t-1) again for t>0, simply swap pointers on device side...

			// calculate and print the gauge quality
			generateGaugeQuality<<<numBlocks*2,32>>>(dUtUp, dGff, dA );
			printGaugeQuality<<<1,1>>>(dGff, dA);


			float orParameter = 1.7;

			Chronotimer kernelTimer;
			kernelTimer.reset();
			kernelTimer.start();
			for( int i = 0; i < 15000; i++ )
			{
				orStep<<<numBlocks,threadsPerBlock>>>(dUtUp, dUtDw, dNnt, 0, orParameter );
				orStep<<<numBlocks,threadsPerBlock>>>(dUtUp, dUtDw, dNnt, 1, orParameter );

				if( i % 100 == 0 )
				{
					projectSU3<<<numBlocks*2,32>>>( dUtUp );
					projectSU3<<<numBlocks*2,32>>>( dUtDw );
					generateGaugeQuality<<<numBlocks*2,32>>>(dUtUp, dGff, dA );
					printGaugeQuality<<<1,1>>>(dGff, dA);
					cout << "time: " << kernelTimer.getTime() << " s"<< endl;
				}
			}
			hipDeviceSynchronize();
			kernelTimer.stop();
			cout << "kernel time for timeslice: " << kernelTimer.getTime() << " s"<< endl;
			totalKernelTime += kernelTimer.getTime();
			// copy back TODO: copying back timeslice t is not necessary (only in the end)
			hipMemcpy( &U[t*timesliceArraySize], dUtUp, timesliceArraySize*sizeof(Real), hipMemcpyDeviceToHost );
			hipMemcpy( &U[tDw*timesliceArraySize], dUtDw, timesliceArraySize*sizeof(Real), hipMemcpyDeviceToHost );
		}


		filename << ".gf";
		bool saveOk = lf.save( s, filename.str(), U );
		if( !saveOk )
		{
			cout << "Error while writing." << endl;
			break;
		}
		else
		{
			cout << "File written." << endl;
		}

	}

	allTimer.stop();
	cout << "total time: " << allTimer.getTime() << " s" << endl;
	cout << "total kernel time: " << totalKernelTime << " s" << endl;
	cout << (double)((long)2253*(long)s.getLatticeSize()*(long)5000)/totalKernelTime/1.0e9 << " GFlops at "
			<< (double)((long)192*(long)s.getLatticeSize()*(long)(5000)*(long)sizeof(Real))/totalKernelTime/1.0e9 << "GB/s memory throughput." << endl;


}
