#include "hip/hip_runtime.h"
/************************************************************************
 *
 *  Copyright 2012 Mario Schroeck, Hannes Vogt
 *
 *  This file is part of cuLGT.
 *
 *  cuLGT is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  any later version.
 *
 *  cuLGT is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with cuLGT.  If not, see <http://www.gnu.org/licenses/>.
 *
 ************************************************************************
 * 
 * This class contains the kernels and wrappers called
 * by MultiGPU_MPI_Communicator.hxx.
 * 
 * kernels as class members are not supported (even static): 
 * wrap the kernel calls and hide the kernels in namespace.
 * 
 */

#include "../../kernel_launch_bounds.h"
#include "../../GlobalConstants.h"
#include "../../GaugeFixingSubgroupStep.hxx"
#include "../../algorithms/OrUpdate.hxx"
#include "../../algorithms/MicroUpdate.hxx"
#include "../../algorithms/SaUpdate.hxx"
#include "../../algorithms/RandomUpdate.hxx"
#include "../../../lattice/access_pattern/StandardPattern.hxx"
#include "../../../lattice/access_pattern/GpuPatternTimesliceParityPriority.hxx"
#include "../../../lattice/access_pattern/GpuPatternParityPriority.hxx"
#include "../../../lattice/SiteCoord.hxx"
#include "../../../lattice/SiteIndex.hxx"
#include "../../../lattice/Link.hxx"
#include "../../../lattice/SU3.hxx"
#include "../../../lattice/Matrix.hxx"
#include "../../../lattice/rng/PhiloxWrapper.hxx"

#include "./MultiGPU_MPI_LandauKernelsSU3.h"


// kernels:
namespace MPILKSU3
{

template<class Algorithm> inline __device__ void applyOneTimeslice( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, Algorithm algorithm  )
{
	typedef GpuPatternParityPriority< SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> GpuIndex;
	typedef Link<GpuIndex,SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> TLinkIndex;

	const lat_coord_t size[Ndim] = {1,Nx,Ny,Nz};
	SiteIndex<4,FULL_SPLIT> s(size);
// 	SiteIndex<4,FULL_SPLIT> s( DEVICE_CONSTANTS::SIZE_TIMESLICE );
	
	s.nn = nnt;

	const bool updown = threadIdx.x / (NSB*4);
	const short mu = (threadIdx.x % (NSB*4)) / NSB;
	const short id = (threadIdx.x % (NSB*4)) % NSB;

	int site = blockIdx.x * blockDim.x/8 + id;
	if( parity == 1 ) site += s.getLatticeSize()/2;

	s.setLatticeIndex( site );

	Real* U;
	
	if( updown==1 )
	{
		if( mu!=0 )
		{
			s.setNeighbour(mu,false);
			U=UtUp;
		}
		else
		{
			U=UtDw;
		}
	}
	else
		U=UtUp;

	Matrix<Complex<Real>,Nc> locMat;
	SU3<Matrix<Complex<Real>,Nc> > locU(locMat);

	TLinkIndex link( U, s, mu );

	SU3<TLinkIndex> globU( link );

	// make link local
	locU.assignWithoutThirdLine(globU);
	locU.reconstructThirdLine();


	GaugeFixingSubgroupStep<SU3<Matrix<Complex<Real>,Nc> >, Algorithm, LANDAU> subgroupStep( &locU, algorithm, id, mu, updown );

	// do the subgroup iteration
	SU3<Matrix<Complex<Real>,Nc> >::perSubgroup( subgroupStep );

	// copy link back
	globU.assignWithoutThirdLine(locU);
}

__global__ void generateGaugeQualityPerSite( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, double *dGff, double *dA )
{
	typedef GpuPatternParityPriority< SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> GpuIndex;
	typedef Link<GpuIndex,SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> TLinkIndex;

	const lat_coord_t size[Ndim] = {1,Nx,Ny,Nz};
	SiteIndex<4,FULL_SPLIT> s(size);
// 	SiteIndex<4,FULL_SPLIT> s( DEVICE_CONSTANTS::SIZE_TIMESLICE);
	
	s.nn = nnt;
	
	int site = blockIdx.x * blockDim.x + threadIdx.x;
	int resid = site;
	if( parity == 1 ) site += Nx*Ny*Nz/2;
// 	if( site >= Nx*Ny*Nz ) return; //important in case Nx^3 is not power of 2

	Matrix<Complex<Real>,Nc> locMatSum;
	SU3<Matrix<Complex<Real>,Nc> > Sum(locMatSum);
	Sum.zero();
	double result = 0;


	for( int mu = 0; mu < 4; mu++ )
	{
		s.setLatticeIndex( site );
		Matrix<Complex<Real>,Nc> locMat;
		SU3<Matrix<Complex<Real>,Nc> > temp(locMat);
		TLinkIndex linkUp( UtUp, s, mu );
		SU3<TLinkIndex> globUp( linkUp );
		
		temp.assignWithoutThirdLine( globUp );
		temp.reconstructThirdLine();
		result += temp.trace().x;
		Sum += temp;
		
		if( mu==0 )
		{
			TLinkIndex linkDw( UtDw, s, mu );
			SU3<TLinkIndex> globDw( linkDw );
			temp.assignWithoutThirdLine( globDw );
			temp.reconstructThirdLine();
		}
		else
		{
			s.setNeighbour(mu,false);
			TLinkIndex linkDw( UtUp, s, mu );
			SU3<TLinkIndex> globDw( linkDw );
			temp.assignWithoutThirdLine( globDw );
			temp.reconstructThirdLine();
		}
		Sum -= temp;
	}
	dGff[resid] = result;

	Sum -= Sum.trace()/Real(3.);

	Matrix<Complex<Real>,Nc> locMatSumHerm;
	SU3<Matrix<Complex<Real>,Nc> > SumHerm(locMatSumHerm);
	SumHerm = Sum;
	SumHerm.hermitian();

	Sum -= SumHerm;

	double prec = 0;
	for( int i = 0; i < 3; i++ )
	{
		for( int j = 0; j < 3; j++ )
		{
			prec += Sum.get(i,j).abs_squared();
		}
	}

	dA[resid] = prec;

}



__global__ void __launch_bounds__(8*NSB,OR_MINBLOCKS) orStep( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, float orParameter )
{
	OrUpdate overrelax( orParameter );
	applyOneTimeslice( UtUp, UtDw, nnt, parity, overrelax  );
}

__global__ void __launch_bounds__(8*NSB,MS_MINBLOCKS) microStep( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity )
{
	MicroUpdate micro;
	applyOneTimeslice( UtUp, UtDw, nnt, parity, micro );
}

__global__ void __launch_bounds__(8*NSB,SA_MINBLOCKS)  saStep( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, float temperature, int rngSeed, int rngCounter )
{
	PhiloxWrapper rng( blockIdx.x * blockDim.x + threadIdx.x, rngSeed, rngCounter );
	SaUpdate sa( temperature, &rng );
	applyOneTimeslice( UtUp, UtDw, nnt, parity, sa );
}

__global__ void randomTrafo( Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, int rngSeed, int rngCounter )
{
	PhiloxWrapper rng( blockIdx.x * blockDim.x + threadIdx.x, rngSeed, rngCounter );
	RandomUpdate random( &rng );
	applyOneTimeslice( UtUp, UtDw, nnt, parity, random );
}

__global__ void projectSU3( Real* Ut )
{
	typedef GpuPatternParityPriority< SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> GpuIndex;
	typedef Link<GpuIndex,SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> TLinkIndex;

	const lat_coord_t size[Ndim] = {1,Nx,Ny,Nz};
	SiteIndex<4,FULL_SPLIT> s(size);
	
	int site = blockIdx.x * blockDim.x + threadIdx.x;

	s.setLatticeIndex( site );

	for( int mu = 0; mu < 4; mu++ )
	{
		TLinkIndex linkUp( Ut, s, mu );
		SU3<TLinkIndex> globUp( linkUp );

		globUp.projectSU3(); // IMPORTANT: Currently this kernel is used for reconstructing third line in the end. Be aware of this when changing something.
	}
}

__global__ void setHot( Real* Ut, int rngSeed, int rngCounter )
{
	typedef GpuPatternParityPriority< SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> GpuIndex;
	typedef Link<GpuIndex,SiteIndex<Ndim,FULL_SPLIT>,Ndim,Nc> TLinkIndex;

	const lat_coord_t size[Ndim] = {1,Nx,Ny,Nz};
	SiteIndex<4,FULL_SPLIT> s(size);
	int site = blockIdx.x * blockDim.x + threadIdx.x;
	s.setLatticeIndex( site );
	
	PhiloxWrapper rng( site, rngSeed, rngCounter );

	Quaternion<Real> q;
	
	for( int mu = 0; mu < 4; mu++ )
	{
		TLinkIndex linkUp( Ut, s, mu );
		SU3<TLinkIndex> globUp( linkUp );

		Matrix<Complex<Real>,Nc> locMat;
		SU3<Matrix<Complex<Real>,Nc> > locU(locMat);

		locU.identity();
		
		for( int i=0; i<2; i++ )
			for( int j=i+1; j<3; j++ )
			{
				q[0] = rng.rand()*2.0-1.0;
				q[1] = rng.rand()*2.0-1.0;
				q[2] = rng.rand()*2.0-1.0;
				q[3] = rng.rand()*2.0-1.0;
				
				q.projectSU2();
				locU.rightSubgroupMult( i, j, &q );
			}
 		globUp = locU;
	}
}

}



// wrappers:
// constructor
MultiGPU_MPI_LandauKernelsSU3::MultiGPU_MPI_LandauKernelsSU3()
{
}

// TODO call this function in main or somewhere appropriate
void MultiGPU_MPI_LandauKernelsSU3::initCacheConfig()
{
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( MPILKSU3::generateGaugeQualityPerSite), hipFuncCachePreferL1 );
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( MPILKSU3::randomTrafo), hipFuncCachePreferL1 );
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( MPILKSU3::orStep), hipFuncCachePreferL1 );
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( MPILKSU3::microStep), hipFuncCachePreferL1 );
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( MPILKSU3::saStep), hipFuncCachePreferL1 );
	hipFuncSetCacheConfig(reinterpret_cast<const void*>( MPILKSU3::projectSU3), hipFuncCachePreferL1 );
}


void MultiGPU_MPI_LandauKernelsSU3::applyOneTimeslice( int a, int b, hipStream_t stream, Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, MultiGPU_MPI_AlgorithmOptions algoOptions )
{
	switch( algoOptions.getAlgorithm() )
	{
	case OR:
		MPILKSU3::orStep<<<a,b,0,stream>>>( UtUp, UtDw, nnt, parity, algoOptions.getOrParameter() );
		break;
	case MS:
		MPILKSU3::microStep<<<a,b,0,stream>>>( UtUp, UtDw, nnt, parity );
		break;
	case SA:
		MPILKSU3::saStep<<<a,b,0,stream>>>( UtUp, UtDw, nnt, parity, algoOptions.getTemperature(), PhiloxWrapper::getNextCounter(), algoOptions.getSeed() );
		break;
	case RT:
		MPILKSU3::randomTrafo<<<a,b,0,stream>>>( UtUp, UtDw, nnt, parity, PhiloxWrapper::getNextCounter(), algoOptions.getSeed() );
		break;
	default:
		printf("Algorithm type not set to a known value [MultiGPU_MPI_AlgorithmOptions::setAlgorithm(enum AlgoType)]. Exiting\n");
		exit(1);
	}
}

void MultiGPU_MPI_LandauKernelsSU3::projectSU3( int a, int b, hipStream_t stream, Real* Ut )
{
	MPILKSU3::projectSU3<<<a,b,0,stream>>>( Ut );
}

void MultiGPU_MPI_LandauKernelsSU3::setHot( int a, int b, hipStream_t stream, Real* Ut, MultiGPU_MPI_AlgorithmOptions algoOptions )
{
	
	MPILKSU3::setHot<<<a,b,0,stream>>>( Ut, PhiloxWrapper::getNextCounter(), algoOptions.getSeed() );
}

void MultiGPU_MPI_LandauKernelsSU3::generateGaugeQualityPerSite( int a, int b, hipStream_t stream, Real* UtUp, Real* UtDw, lat_index_t* nnt, bool parity, double *dGff, double *dA )
{
	MPILKSU3::generateGaugeQualityPerSite<<<a,b,0,stream>>>( UtUp, UtDw, nnt, parity, dGff, dA );
}






