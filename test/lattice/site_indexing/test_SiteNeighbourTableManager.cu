#include "hip/hip_runtime.h"

#include "gmock/gmock.h"
#include "lattice/site_indexing/SiteIndex.h"
#include "lattice/site_indexing/SiteNeighbourTableManager.h"
#include "cudacommon/DeviceCommunicator.h"

using namespace testing;
using namespace culgt;


class ASiteNeighbourTableManager: public Test
{
public:
	typedef SiteIndex<4,NO_SPLIT> MySite;
	LatticeDimension<4> dim;

	ASiteNeighbourTableManager() : dim(4,4,4,4)
	{

	}
};

TEST_F( ASiteNeighbourTableManager, IsNotAvailableIfNotCreated )
{
	bool result = SiteNeighbourTableManager<MySite>::isAvailable( dim );

	ASSERT_FALSE( result );
}

TEST_F( ASiteNeighbourTableManager, IsAvailableIfCreated )
{
	SiteNeighbourTableManager<MySite>::generate( dim );

	bool result = SiteNeighbourTableManager<MySite>::isAvailable( dim );

	ASSERT_TRUE( result );
}

TEST_F( ASiteNeighbourTableManager, NeighbourIndexIsCorrect )
{
	MySite site( dim, SiteNeighbourTableManager<MySite>::getHostPointer( dim ) );

	site.setIndex( 0 );
	site.setNeighbour( 3, true );

	ASSERT_EQ( 1, site.getIndex() );
}

__global__ void kernelTestNeighbourTable( LatticeDimension<4> dim, lat_index_t* nn, lat_index_t* var )
{
	typedef SiteIndex<4,NO_SPLIT> MySite;
	MySite site( dim, nn );

	site.setIndex( 0 );
	site.setNeighbour( 3, true );

	var[0] = site.getIndex();
}

TEST( ASiteNeighbourTableManagerOnDevice, NeighbourIndexIsCorrect )
{
	typedef SiteIndex<4,NO_SPLIT> MySite;
	LatticeDimension<4> dim(8,8,8,8);

	lat_index_t* deviceVar;
	hipMalloc( &deviceVar, sizeof(lat_index_t) );

	kernelTestNeighbourTable<<<1,1>>>( dim, SiteNeighbourTableManager<MySite>::getDevicePointer( dim ), deviceVar );
	CUDA_LAST_ERROR( "test kernel" );

	ASSERT_EQ( 1, DeviceCommunicator<lat_index_t>::getValue( deviceVar, 0 ) );
}
